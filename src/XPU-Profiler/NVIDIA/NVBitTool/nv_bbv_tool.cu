#include "hip/hip_runtime.h"
/* BEGIN_LEGAL
 * Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 * The MIT License (MIT)
 *
 * Copyright (c) 2024, National University of Singapore
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 END_LEGAL */

#include <algorithm>
#include <assert.h>
#include <bitset>
#include <inttypes.h>
#include <iostream>
#include <iterator>
#include <map>
#include <sstream>
#include <fstream>
#include <stdint.h>
#include <stdio.h>
#include <string>
#include <sys/stat.h>
#include <unistd.h>
#include <unordered_set>
#include <unordered_map>
#include <vector>
/* every tool needs to include this once */
#include "nvbit_tool.h"

/* nvbit interface file */
#include "nvbit.h"

/* for channel */
#include "utils/channel.hpp"

/* contains definition of the inst_trace_t structure */
#include "common.h"

/* nvbit interface file */
#include "NVBitShim.h"

#define MAX_THREADS 512

/* Channel used to communicate from GPU to CPU receiving thread */
#define CHANNEL_SIZE (1l << 24)
static __managed__ ChannelDev channel_dev;
static ChannelHost channel_host;

cpu_on_gpu_fini_ptr_t cpu_on_gpu_fini = 0;
cpu_on_gpu_init_ptr_t cpu_on_gpu_init = 0;
cpu_on_kernel_complete_ptr_t cpu_on_kernel_complete = 0;

/* receiving thread and its control variables */
pthread_t recv_thread;
volatile bool recv_thread_started = false;
volatile bool recv_thread_receiving = false;

/* skip flag used to avoid re-entry on the nvbit_callback when issuing
 * flush_channel kernel call */
bool mangled = false;
bool skip_flag = false;

/* global control variables for this tool */
uint32_t instr_begin_interval = 0;
uint32_t instr_end_interval = UINT32_MAX;
int verbose = 0;
int exclude_pred_off = 1;
int active_from_start = 1;
/* used to select region of interest when active from start is 0 */
bool active_region = true;

/* Should we terminate the program once we are done tracing? */
int terminate_after_limit_number_of_kernels_reached = 0;
int user_defined_directory = 0;

uint64_t kernel_counter = 1;
int max_warpid = 0;
std::vector<std::tuple<std::string, uint64_t>> region_boundary; 
std::map<std::string, uint64_t> curr_kernel_call; // kernel_name->call_num
std::string curr_kernel;
typedef struct
{
  std::map<std::tuple<uint64_t, uint32_t>, uint32_t> curr_bbv;  // store current bbv
  uint64_t insn_count;
} thread_data_t;
thread_data_t td[MAX_THREADS];
uint64_t curr_bbvid = 1;
std::map<std::tuple<uint64_t,uint32_t>, uint64_t> bbvids; // store (block-addr,inst-count)->bbid

/* opcode to id map and reverse map  */
std::map<std::string, int> opcode_to_id_map;
std::map<int, std::string> id_to_opcode_map;

std::string cwd = getcwd(NULL,0);
std::string tidstr = std::to_string(getpid());
std::string traces_location = cwd + "/BasicBlocks." + tidstr;
std::string stats_location = traces_location + "/stats.csv";
std::string global_bbv_location = traces_location + "/global.bbv";
std::string thread_bbv_location = traces_location + "/thread.bbv";

/* kernel instruction counter, updated by the GPU */
uint64_t dynamic_kernel_limit_start = 0; // 0 means start from the first kernel
uint64_t dynamic_kernel_limit_end = 0; // 0 means no limit

static FILE *statsFile = NULL;
std::ofstream globalBbvFile;
std::ofstream threadBbvFile;
static uint64_t kernelid = 1;
static bool first_call = true;

unsigned old_total_insts = 0;
unsigned old_total_reported_insts = 0;

void nvbit_at_init() {
  setenv("CUDA_MANAGED_FORCE_DEVICE_ALLOC", "1", 1);
  GET_VAR_INT(
      instr_begin_interval, "INSTR_BEGIN", 0,
      "Beginning of the instruction interval where to apply instrumentation");
  GET_VAR_INT(instr_end_interval, "INSTR_END", UINT32_MAX,
      "End of the instruction interval where to apply instrumentation");
  GET_VAR_INT(exclude_pred_off, "EXCLUDE_PRED_OFF", 1,
      "Exclude predicated off instruction from count");
  GET_VAR_INT(dynamic_kernel_limit_end, "DYNAMIC_KERNEL_LIMIT_END", 0,
      "Limit of the number kernel to be printed, 0 means no limit");
  GET_VAR_INT(dynamic_kernel_limit_start, "DYNAMIC_KERNEL_LIMIT_START", 0,
      "start to report kernel from this kernel id, 0 means starts from "
      "the beginning, i.e. first kernel");
  GET_VAR_INT(
      active_from_start, "ACTIVE_FROM_START", 1,
      "Start instruction tracing from start or wait for hipProfilerStart "
      "and hipProfilerStop. If set to 0, DYNAMIC_KERNEL_LIMIT options have no effect");
  GET_VAR_INT(verbose, "TOOL_VERBOSE", 0, "Enable verbosity inside the tool");
  GET_VAR_INT(terminate_after_limit_number_of_kernels_reached, "TERMINATE_UPON_LIMIT", 0, 
      "Stop the process once the current kernel > DYNAMIC_KERNEL_LIMIT_END");
  GET_VAR_INT(user_defined_directory, "USER_DEFINED_DIRECTORY", 0, "Uses the user defined "
      "BBV_DIR path environment");

  if (active_from_start == 0) {
    active_region = false;
  }
}

/* Set used to avoid re-instrumenting the same functions multiple times */
std::unordered_set<hipFunction_t> already_instrumented;

/* instrument each memory instruction adding a call to the above instrumentation
 * function */
void instrument_function_if_needed(hipCtx_t ctx, hipFunction_t func) {

  std::vector<hipFunction_t> related_functions =
    nvbit_get_related_functions(ctx, func);

  /* add kernel itself to the related function vector */
  related_functions.push_back(func);

  /* iterate on function */
  for (auto f : related_functions) {
    /* "recording" function was instrumented, if set insertion failed
     * we have already encountered this function */
    if (!already_instrumented.insert(f).second) {
      continue;
    }

    const std::vector<Instr *> &instrs = nvbit_get_instrs(ctx, f);
    if (verbose) {
      printf("Inspecting function %s at address 0x%lx\n", nvbit_get_func_name(ctx, f), nvbit_get_func_addr(f));
    }

    const CFG_t &cfg = nvbit_get_CFG(ctx, f);
    for (auto &bb : cfg.bbs) {
      Instr *instr = bb->instrs[0];
      auto addr = instr->getOffset();
      auto num_insn = bb->instrs.size();

      if (opcode_to_id_map.find(instr->getOpcode()) == opcode_to_id_map.end()) {
        int opcode_id = opcode_to_id_map.size();
        opcode_to_id_map[instr->getOpcode()] = opcode_id;
        id_to_opcode_map[opcode_id] = instr->getOpcode();
      }

      int opcode_id = opcode_to_id_map[instr->getOpcode()];

      /* insert call to the instrumentation function with its
       * arguments */
      nvbit_insert_call(instr, "instrument_inst", IPOINT_BEFORE);

      /* pass predicate value */
      nvbit_add_call_arg_guard_pred_val(instr);

      /* send opcode and pc */
      nvbit_add_call_arg_const_val32(instr, opcode_id);
      nvbit_add_call_arg_const_val32(instr, (int)instr->getOffset());

      /* add pointer to channel_dev and other counters*/
      nvbit_add_call_arg_const_val64(instr, (uint64_t)&channel_dev);
      nvbit_add_call_arg_const_val64(instr,
          (uint64_t)&total_dynamic_instr_counter);
      nvbit_add_call_arg_const_val64(instr,
          (uint64_t)&reported_dynamic_instr_counter);
      nvbit_add_call_arg_const_val64(instr, (uint64_t)&stop_report);
      nvbit_add_call_arg_const_val32(instr, (uint32_t)bb->instrs.size());
    }
  }
}

__global__ void flush_channel() {
  /* push memory access with negative cta id to communicate the kernel is
   * completed */
  inst_trace_t ma;
  ma.cta_id_x = -1;
  channel_dev.push(&ma, sizeof(inst_trace_t));

  /* flush channel */
  channel_dev.flush();
}

void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
    const char *name, void *params, hipError_t *pStatus) {

  if (skip_flag)
    return;

  if (first_call == true) {

    first_call = false;

    if (active_from_start && !dynamic_kernel_limit_start || dynamic_kernel_limit_start == 1)
      active_region = true;
    else {
      if (active_from_start)
        active_region = false;
    }

    if(user_defined_directory == 1)
    {
      std::string usr_dir = std::getenv("BBV_DIR");
      std::string temp_traces_location = usr_dir + "." + tidstr;
      std::string temp_stats_location = temp_traces_location + "/stats.csv";
      std::string temp_global_bbv_location = temp_traces_location + "/global.bbv";
      std::string temp_thread_bbv_location = temp_traces_location + "/thread.bbv";
      traces_location = temp_traces_location;
      stats_location = temp_stats_location;
      global_bbv_location = temp_global_bbv_location;
      thread_bbv_location = temp_thread_bbv_location;
      std::cerr << "[XPU_TRACER][" << __FUNCTION__ << "Traces location is " << traces_location << std::endl;
    }

    if (mkdir(traces_location.c_str(), S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH) == -1) {
      if (errno == EEXIST) {
        std::cerr << "[XPU_TRACER][" << __FUNCTION__ << "] Directory " << traces_location << " exists" << std::endl;
      } else {
        std::cerr << "[XPU_TRACER][" << __FUNCTION__ << "] Cannot create directory " << traces_location << ". Error: " << strerror(errno) << std::endl;
        return;
      }
    }

    statsFile = fopen(stats_location.c_str(), "w+");
    fprintf(statsFile,
        "kernel id, kernel mangled name, grid_dimX, grid_dimY, grid_dimZ, "
        "#blocks, block_dimX, block_dimY, block_dimZ, #threads, "
        "total_insts, total_reported_insts\n");
    fclose(statsFile);
    if (!globalBbvFile.is_open()) {
      globalBbvFile.open(global_bbv_location, std::ios_base::out | std::ios_base::trunc);
      std::cout << "[XPU_TRACER][" << __FUNCTION__ << "] Opened " << global_bbv_location << std::endl;
      globalBbvFile << "M: SYS_init 1" << std::endl;
    }
    if (!threadBbvFile.is_open()) {
      threadBbvFile.open(thread_bbv_location, std::ios_base::out | std::ios_base::trunc);
      std::cout << "[XPU_TRACER][" << __FUNCTION__ << "] Opened " << thread_bbv_location << std::endl;
      threadBbvFile << "M: SYS_init 1" << std::endl;
    }
  }

  if (cbid == API_CUDA_cuLaunchKernel_ptsz ||
      cbid == API_CUDA_cuLaunchKernel) {
    cuLaunchKernel_params *p = (cuLaunchKernel_params *)params;

    if (!is_exit) {

      if (active_from_start && dynamic_kernel_limit_start && kernelid == dynamic_kernel_limit_start)
        active_region = true;

      if (terminate_after_limit_number_of_kernels_reached && dynamic_kernel_limit_end != 0 && kernelid > dynamic_kernel_limit_end)
      {
        exit(0);
      }

      const char* kernel_name = nvbit_get_func_name(ctx, p->f, true);
      std::string kname(kernel_name);
      if (curr_kernel_call.find(kname) == curr_kernel_call.end()) {
        curr_kernel_call[kname] = 1;
      }
      else {
        curr_kernel_call[kname]++;
      }
      region_boundary.push_back(std::make_pair(kname, curr_kernel_call[kname]));
      curr_kernel = kname;
      int nregs;
      CUDA_SAFECALL(
          hipFuncGetAttribute(&nregs, HIP_FUNC_ATTRIBUTE_NUM_REGS, p->f));

      int shmem_static_nbytes;
      CUDA_SAFECALL(hipFuncGetAttribute(
            &shmem_static_nbytes, HIP_FUNC_ATTRIBUTE_SHARED_SIZE_BYTES, p->f));

      int binary_version;
      CUDA_SAFECALL(hipFuncGetAttribute(&binary_version,
            HIP_FUNC_ATTRIBUTE_BINARY_VERSION, p->f));

      instrument_function_if_needed(ctx, p->f);

      if (active_region) {
        nvbit_enable_instrumented(ctx, p->f, true);
        stop_report = false;
      } else {
        nvbit_enable_instrumented(ctx, p->f, false);
        stop_report = true;
      }

      statsFile = fopen(stats_location.c_str(), "a");
      unsigned blocks = p->gridDimX * p->gridDimY * p->gridDimZ;
      unsigned threads = p->blockDimX * p->blockDimY * p->blockDimZ;

      fprintf(statsFile, "%ld, %s, %d, %d, %d, %d, %d, %d, %d, %d, ", kernelid,
          nvbit_get_func_name(ctx, p->f, true), p->gridDimX, p->gridDimY,
          p->gridDimZ, blocks, p->blockDimX, p->blockDimY, p->blockDimZ,
          threads);

      fclose(statsFile);
      kernelid++;
      recv_thread_receiving = true;

    } else {
      /* make sure current kernel is completed */
      if (verbose)
        std::cerr << "[XPU_TRACER][" << __FUNCTION__ << "] Kernel completed." << std::endl;
      hipDeviceSynchronize();
      assert(hipGetLastError() == hipSuccess);

      /* make sure we prevent re-entry on the nvbit_callback when issuing
       * the flush_channel kernel */
      skip_flag = true;

      /* issue flush of channel so we are sure all the memory accesses
       * have been pushed */
      flush_channel<<<1, 1>>>();
      hipDeviceSynchronize();
      assert(hipGetLastError() == hipSuccess);
      if (verbose)
        std::cerr << "Kernel exit counter: " << kernel_counter << std::endl;
      kernel_counter++;
      /* unset the skip flag */
      skip_flag = false;

      if(cpu_on_kernel_complete)
      {
        if(verbose) printf("Event: on_complete %lx\n", (uint64_t) cpu_on_kernel_complete);
        (*cpu_on_kernel_complete)(curr_kernel.c_str());
        if(verbose) printf("\n");
      }

      /* wait here until the receiving thread has not finished with the
       * current kernel */
      while (recv_thread_receiving) {
        pthread_yield();
      }

      if (verbose) {
        std::cerr  << "[XPU_TRACER][" << __FUNCTION__ << "] Ending region for kernel: " << curr_kernel << " call: " << curr_kernel_call[curr_kernel] << std::endl;
        std::cerr  << "[XPU_TRACER][" << __FUNCTION__ << "] Dynamic instruction count on GPU: " << total_dynamic_instr_counter << std::endl;
      }
      for (auto i = 0; i <= max_warpid; i++) {
        for (auto &v: td[i].curr_bbv) {
          if (bbvids.find(v.first) == bbvids.end()) {
            bbvids[v.first] = curr_bbvid;
            curr_bbvid++;
          }
        }
      }
      globalBbvFile << "# Slice ending at kernel: " << curr_kernel << " call: " << curr_kernel_call[curr_kernel] << std::endl;
      threadBbvFile << "# Slice ending at kernel: " << curr_kernel << " call: " << curr_kernel_call[curr_kernel] << std::endl;

      std::map<uint64_t, uint64_t> bbid_global;
      for (auto i = 0; i <= max_warpid; i++) {
        if (td[i].curr_bbv.empty())
          continue;
        threadBbvFile << "tid" << i << ": T";
        for (auto &m: td[i].curr_bbv) {
          auto bb = bbvids[m.first];
          bbid_global[bb] += m.second;
          threadBbvFile << ":" << bb << ":" << m.second << " ";
        }
        threadBbvFile << std::endl;
      }
      globalBbvFile << "T";
      for (auto& el:bbid_global) {
        globalBbvFile << ":" << el.first << ":" << el.second << " ";
      }
      globalBbvFile << std::endl;

      globalBbvFile << "M: " << curr_kernel << " " << curr_kernel_call[curr_kernel] << std::endl;
      threadBbvFile << "M: " << curr_kernel << " " << curr_kernel_call[curr_kernel] << std::endl;

      for (auto tid = 0; tid <= max_warpid; tid++) {
        if (!td[tid].curr_bbv.empty()) {
          td[tid].curr_bbv.clear();
        }
      }

      unsigned total_insts_per_kernel = total_dynamic_instr_counter - old_total_insts;
      old_total_insts = total_dynamic_instr_counter;

      unsigned reported_insts_per_kernel = reported_dynamic_instr_counter - old_total_reported_insts;
      old_total_reported_insts = reported_dynamic_instr_counter;

      statsFile = fopen(stats_location.c_str(), "a");
      fprintf(statsFile, "%d, %d", total_insts_per_kernel,	reported_insts_per_kernel);
      fprintf(statsFile, "\n");
      fclose(statsFile);

      if (active_from_start && dynamic_kernel_limit_end && kernelid > dynamic_kernel_limit_end)
        active_region = false;
    }
  } else if (cbid == API_CUDA_cuProfilerStart && is_exit) {
    if (!active_from_start) {
      active_region = true;
    }
  } else if (cbid == API_CUDA_cuProfilerStop && is_exit) {
    if (!active_from_start) {
      active_region = false;
    }
  }
}

void *recv_thread_fun(void *) {
  bool new_kernel = true;
  char *recv_buffer = (char *)malloc(CHANNEL_SIZE);
  while (recv_thread_started) {
    uint32_t num_recv_bytes = 0;

    if (recv_thread_receiving &&
        (num_recv_bytes = channel_host.recv(recv_buffer, CHANNEL_SIZE)) > 0) {
      if (new_kernel) {
        new_kernel = false;
        if (verbose >= 6) {
          std::cerr << "bb-ins, bb-vpc, warp-id-tb, warp-id-sm, thread-id, sm-id" << std::endl;
        }
      }
      uint32_t num_processed_bytes = 0;
      while (num_processed_bytes < num_recv_bytes) {
        inst_trace_t *ma = (inst_trace_t *)&recv_buffer[num_processed_bytes];

        /* when we get this cta_id_x it means the kernel has completed
         */
        if (ma->cta_id_x == -1) {
          recv_thread_receiving = false;
          new_kernel = true;
          break;
        }
        if (verbose >= 6) {
          std::cerr << ma->num_insn << ", " << ma->vpc << ", " << ma->warpid_tb << ", ";
          std::cerr << ma->warpid_sm << ", " << ma->thread_id << ", " << ma->sm_id << std::endl;
        }

        td[ma->warpid_sm].curr_bbv[std::tuple<uint64_t, uint32_t>(ma->vpc, ma->num_insn)] +=  ma->num_insn;
        td[ma->warpid_sm].insn_count += ma->num_insn;
        if (ma->warpid_sm > max_warpid)
          max_warpid = ma->warpid_sm;
        if (max_warpid >= MAX_THREADS) {
          std::cerr << "[XPU_TRACER][" << __FUNCTION__ << "] Error: Unable to handle the large number of warps." << std::endl;
        }
        num_processed_bytes += sizeof(inst_trace_t);
      }
    }
  }
  free(recv_buffer);
  return NULL;
}

void nvbit_at_ctx_init(hipCtx_t ctx) {
  recv_thread_started = true;
  channel_host.init(0, CHANNEL_SIZE, &channel_dev, NULL);
  pthread_create(&recv_thread, NULL, recv_thread_fun, NULL);
}

void nvbit_at_ctx_term(hipCtx_t ctx) {
  if (recv_thread_started) {
    recv_thread_started = false;
    pthread_join(recv_thread, NULL);
  }
}

void nvbit_at_term() {
  if (verbose)
    printf(" nvbit_at_term() \n");
  if(cpu_on_gpu_fini)
  {
    (*cpu_on_gpu_fini)();
  }
  for (auto tid = 0; tid <= max_warpid; tid++) {
    if (!td[tid].curr_bbv.empty()) {
      std::cerr << "[XPU_TRACER][" << __FUNCTION__ << "] Non-empty BBV for thread/warp " << tid << " at the end of program." << std::endl;
      td[tid].curr_bbv.clear();
    }
  }
  if (threadBbvFile.is_open()) {
    threadBbvFile.close();
  }
  if (globalBbvFile.is_open()) {
    globalBbvFile.close();
  }
}

void NVBitShimRegisterCallbacks(void *ptri, void *ptrc, void * ptrf)
{
  if (verbose)
    printf("NVBitShimRegisterCallbacks: on_init %lx, on_complete %lx, on_fini %lx\n", (uint64_t) ptri, (uint64_t) ptrc, (uint64_t) ptrf);
  cpu_on_gpu_init = (cpu_on_gpu_init_ptr_t) ptri;
  cpu_on_kernel_complete = (cpu_on_kernel_complete_ptr_t) ptrc;
  cpu_on_gpu_fini = (cpu_on_gpu_fini_ptr_t) ptrf;
  if(cpu_on_gpu_init)
  {
    if (verbose)
      printf("calling  (*cpu_on_gpu_init)()\n");
    (*cpu_on_gpu_init)();
  }
}
