#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <cstdarg>
#include <stdint.h>
#include <stdio.h>

#include "utils/utils.h"

/* for channel */
#include "utils/channel.hpp"

/* contains definition of the inst_trace_t structure */
#include "common.h"

/* Instrumentation function that we want to inject, please note the use of
 *  extern "C" __device__ __noinline__
 *    To prevent "dead"-code elimination by the compiler.
 */
extern "C" __device__ __noinline__ void instrument_inst(
    int pred, int opcode_id, int32_t vpc, 
    uint64_t pchannel_dev, uint64_t ptotal_dynamic_instr_counter,
    uint64_t preported_dynamic_instr_counter, uint64_t pstop_report, 
		uint32_t pnum_insn) {

  const int active_mask = __ballot_sync(__activemask(), 1);
  const int predicate_mask = __ballot_sync(__activemask(), pred);
  const int laneid = get_laneid();
  const int first_laneid = __ffs(active_mask) - 1;

  if ((*((bool *)pstop_report))) {
    if (first_laneid == laneid) {
      atomicAdd((unsigned long long *)ptotal_dynamic_instr_counter, pnum_insn);
      return;
    }
  }

  inst_trace_t ma;

  int4 cta = get_ctaid();
  int uniqe_threadId = threadIdx.z * blockDim.y * blockDim.x +
                       threadIdx.y * blockDim.x + threadIdx.x;
	ma.warpid_tb = uniqe_threadId / 32;
  ma.warpid_sm = get_warpid();
  ma.cta_id_x = cta.x;
  ma.cta_id_y = cta.y;
  ma.cta_id_z = cta.z;
  ma.opcode_id = opcode_id;
  ma.vpc = vpc;
  ma.num_insn = pnum_insn;
	ma.thread_id = uniqe_threadId;
  ma.active_mask = active_mask;
  ma.predicate_mask = predicate_mask;
  ma.sm_id = get_smid();

  /* first active lane pushes information on the channel */
  if (first_laneid == laneid) {
    ChannelDev *channel_dev = (ChannelDev *)pchannel_dev;
    channel_dev->push(&ma, sizeof(inst_trace_t));
    atomicAdd((unsigned long long *)ptotal_dynamic_instr_counter, pnum_insn);
    atomicAdd((unsigned long long *)preported_dynamic_instr_counter, pnum_insn);
  }
}
