/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <assert.h>
#include <pthread.h>
#include <stdint.h>
#include <stdio.h>
#include <unordered_set>
#include <fstream>
#include <set>
#include <sys/types.h>
#include <unistd.h>
#include <sys/stat.h>


/* every tool needs to include this once */
#include "nvbit_tool.h"

/* nvbit interface file */
#include "nvbit.h"

/* nvbit utility functions */
#include "utils/utils.h"

/* nvbit interface file */
#include "NVBitShim.h"

using namespace std;

cpu_on_gpu_fini_ptr_t cpu_on_gpu_fini = 0;
cpu_on_gpu_init_ptr_t cpu_on_gpu_init = 0;
cpu_on_kernel_complete_ptr_t cpu_on_kernel_complete = 0;

uint32_t kernel_id = 0;

/* global control variables for this tool */
int verbose = 0;
bool mangled = true;
std::string  gpu_perfout = "gpu_perfout.txt";
std::string  gpu_outdir = "./gpuperfdir";
std::ofstream perf_fs;

uint64_t myrdtsc(){
    unsigned int lo,hi;
    __asm__ __volatile__ ("rdtsc" : "=a" (lo), "=d" (hi));
    return ((uint64_t)hi << 32) | lo;
}
uint64_t                _completeCounter = 0;
uint64_t								_runCounter = 0;

/* used to select region of insterest when active from start is off */
bool active_region = true;

/* a pthread mutex, used to prevent multiple kernels to run concurrently and
 * therefore to "corrupt" the counter variable */
pthread_mutex_t mutex;

/* nvbit_at_init() is executed as soon as the nvbit tool is loaded. We typically
 * do initializations in this call. In this case for instance we get some
 * environment variables values which we use as input arguments to the tool */
void nvbit_at_init() {
    /* just make sure all managed variables are allocated on GPU */
    setenv("CUDA_MANAGED_FORCE_DEVICE_ALLOC", "1", 1);

    /* we get some environment variables that are going to be use to selectively
     * instrument (within a interval of kernel indexes and instructions). By
     * default we instrument everything. */

    GET_VAR_INT(verbose, "TOOL_VERBOSE", 0, "Enable verbosity inside the tool");
    std::string pad(100, '-');
    printf("%s\n", pad.c_str());

    GET_VAR_STR(gpu_perfout, "TOOL_GPU_PERFOUT", "Output xpu_event RDTSC here");
    printf("%s\n", pad.c_str());

    GET_VAR_STR(gpu_outdir, "TOOL_GPU_OUTDIR", "Output dirctory");
    printf("%s\n", pad.c_str());

   if(gpu_perfout != "" )
   {
      string fname = gpu_perfout;
      string outdir = gpu_outdir;
      string tidstr = std::to_string(getpid());
      printf("outdir %s\n", outdir.c_str());
      outdir=outdir+"."+tidstr;
      printf("outdir %s\n", outdir.c_str());
      mkdir(outdir.c_str(), S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH);
      string outfile=outdir+"/"+ fname;
      printf("outfile %s\n", outfile.c_str());
      perf_fs.open(outfile);
   }
   // begining of slice/interval 0
   uint64_t init_rdtsc = myrdtsc();
   if(perf_fs.is_open()) perf_fs << "0 GPU_Init : TSC " << init_rdtsc << endl;
}

/* Set used to avoid re-instrumenting the same functions multiple times */
std::unordered_set<hipFunction_t> already_instrumented;

void instrument_function_if_needed(hipCtx_t ctx, hipFunction_t func) {
}

/* This call-back is triggered every time a CUDA driver call is encountered.
 * Here we can look for a particular CUDA driver call by checking at the
 * call back ids  which are defined in tools_cuda_api_meta.h.
 * This call back is triggered bith at entry and at exit of each CUDA driver
 * call, is_exit=0 is entry, is_exit=1 is exit.
 * */
void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
                         const char *name, void *params, hipError_t *pStatus) {
    /* Identify all the possible CUDA launch events */
    if (cbid == API_CUDA_cuLaunch || cbid == API_CUDA_cuLaunchKernel_ptsz ||
        cbid == API_CUDA_cuLaunchGrid || cbid == API_CUDA_cuLaunchGridAsync ||
        cbid == API_CUDA_cuLaunchKernel) {
        /* cast params to cuLaunch_params since if we are here we know these are
         * the right parameters type */
        cuLaunch_params *p = (cuLaunch_params *)params;

        if (!is_exit) {
            /* if we are entering in a kernel launch:
             * 1. Lock the mutex to prevent multiple kernels to run concurrently
             * (overriding the counter) in case the user application does that
             * 2. Instrument the function if needed
             * 3. Select if we want to run the instrumented or original
             * version of the kernel
             * 4. Reset the kernel instruction counter */

            pthread_mutex_lock(&mutex);
            //instrument_function_if_needed(ctx, p->f);
						
            if (active_region) {
                nvbit_enable_instrumented(ctx, p->f, true);
            } else {
                nvbit_enable_instrumented(ctx, p->f, false);
            }
						string kname = nvbit_get_func_name(ctx, p->f, mangled);
						uint64_t onrun_rdtsc = myrdtsc();
						if(perf_fs.is_open()) perf_fs << _runCounter << " OnRun " << kname << " TSC " << onrun_rdtsc << endl;
            ++_runCounter;

            //if(cpu_on_kernel_run)
            //{
              //printf("Event: on_run %lx\n", (uint64_t) cpu_on_kernel_run);
              //(*cpu_on_kernel_run)(nvbit_get_func_name(ctx, p->f, mangled));
            //}
        } else {
            /* if we are exiting a kernel launch:
             * 1. Wait until the kernel is completed using
             * hipDeviceSynchronize()
             * 2. Get number of thread blocks in the kernel
             * 3. Print the thread instruction counters
             * 4. Release the lock*/
            CUDA_SAFECALL(hipDeviceSynchronize());
            string kname = nvbit_get_func_name(ctx, p->f, mangled);
            uint64_t oncomplete_rdtsc = myrdtsc();
            //printf( "kernel %d - %s\n",
            //   kernel_id++, nvbit_get_func_name(ctx, p->f, mangled));
            if(cpu_on_kernel_complete)
            {
              (*cpu_on_kernel_complete)(kname.c_str());
            }
          if(perf_fs.is_open()) perf_fs << _completeCounter << " OnComplete " << kname << " TSC " << oncomplete_rdtsc << endl;
            ++_completeCounter;
            pthread_mutex_unlock(&mutex);
        }
    } else if (cbid == API_CUDA_cuProfilerStart && is_exit) {
            active_region = true;
    } else if (cbid == API_CUDA_cuProfilerStop && is_exit) {
            active_region = false;
    }
}

void nvbit_at_term() {
    printf(" nvbit_at_term() \n");
    uint64_t fini_rdtsc = myrdtsc();
    if(perf_fs.is_open()) perf_fs << _completeCounter << " GPU_Fini : TSC " << fini_rdtsc << endl;
    if(cpu_on_gpu_fini)
    {
      (*cpu_on_gpu_fini)();
    }
}

//void NVBitShimRegisterCallbacks(void * ptrb, void * ptrr, void * ptrc, void * ptrf)
void NVBitShimRegisterCallbacks(void *ptri, void *ptrc, void * ptrf)
{
    printf("NVBitShimRegisterCallbacks: on_init %lx, on_complete %lx, on_fini %lx\n", (uint64_t) ptri, (uint64_t) ptrc, (uint64_t) ptrf);
    cpu_on_gpu_init = (cpu_on_gpu_init_ptr_t) ptri;
    cpu_on_kernel_complete = (cpu_on_kernel_complete_ptr_t) ptrc;
    cpu_on_gpu_fini = (cpu_on_gpu_fini_ptr_t) ptrf;
    if(cpu_on_gpu_init)
    {
      printf("calling  (*cpu_on_gpu_init)()\n");
      (*cpu_on_gpu_init)();
    }
}
